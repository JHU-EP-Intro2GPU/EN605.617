 
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include ormqr_example.cpp 
 *   nvcc -o -fopenmp a.out ormqr_example.o -L/usr/local/cuda/lib64 -lcudart -lcublas -lcusolver
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>


void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}


int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = 3;
    const int lda = m;
    const int ldb = m;
    const int nrhs = 1; // number of right hand side vectors
/*       | 1 2 3 |
 *   A = | 4 5 6 |
 *       | 2 1 1 |
 *
 *   x = (1 1 1)'
 *   b = (6 15 4)'
 */

 

    double A[lda*m] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 3.0, 6.0, 1.0}; 
//    double X[ldb*nrhs] = { 1.0, 1.0, 1.0}; // exact solution
    double B[ldb*nrhs] = { 6.0, 15.0, 4.0}; 
    double XC[ldb*nrhs]; // solution matrix from GPU

    double *d_A = NULL; // linear memory of GPU  
    double *d_tau = NULL; // linear memory of GPU 
    double *d_B  = NULL; 
    int *devInfo = NULL; // info in gpu (device copy)
    double *d_work = NULL;
    int  lwork = 0; 

    int info_gpu = 0;

    const double one = 1;

    printf("A = (matlab base-1)\n");
    printMatrix(m, m, A, lda, "A");
    printf("=====\n");
    printf("B = (matlab base-1)\n");
    printMatrix(m, nrhs, B, ldb, "B");
    printf("=====\n");

// step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    
// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m   , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

 
// step 3: query working space of geqrf and ormqr
    cusolver_status = hipsolverDnDgeqrf_bufferSize(
        cusolverH, 
        m, 
        m, 
        d_A, 
        lda, 
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
 
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute QR factorization
    cusolver_status = hipsolverDnDgeqrf(
        cusolverH, 
        m, 
        m, 
        d_A, 
        lda, 
        d_tau, 
        d_work, 
        lwork, 
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after geqrf: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

// step 5: compute Q^T*B
    cusolver_status= hipsolverDnDormqr(
        cusolverH, 
        HIPBLAS_SIDE_LEFT, 
        HIPBLAS_OP_T,
        m, 
        nrhs, 
        m, 
        d_A, 
        lda,
        d_tau,
        d_B,
        ldb,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

 
    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after ormqr: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

// step 6: compute x = R \ Q^T*B

    cublas_status = hipblasDtrsm(
         cublasH,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N, 
         HIPBLAS_DIAG_NON_UNIT,
         m,
         nrhs,
         &one,
         d_A,
         lda,
         d_B,
         ldb);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(XC, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("X = (matlab base-1)\n");
    printMatrix(m, nrhs, XC, ldb, "X");

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_tau  ) hipFree(d_tau);
    if (d_B    ) hipFree(d_B);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);


    if (cublasH ) hipblasDestroy(cublasH);   
    if (cusolverH) hipsolverDnDestroy(cusolverH);   

    hipDeviceReset();

    return 0;
}
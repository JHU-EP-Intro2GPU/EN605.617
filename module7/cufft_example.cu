#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>

using namespace std;
typedef float2 Complex;


//Found at http://techqa.info/programming/question/36889333/cuda-cufft-2d-example


__global__ void ComplexMUL(Complex *a, Complex *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    a[i].x = a[i].x * b[i].x - a[i].y*b[i].y;
    a[i].y = a[i].x * b[i].y + a[i].y*b[i].x;
}


int main()
{


    int N = 5;
    int SIZE = N*N;


    Complex *fg = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fg[i].x = 1;
        fg[i].y = 0;
    }
    Complex *fig = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fig[i].x = 1; // 
        fig[i].y = 0;
    }
    for (int i = 0; i < N * N; i = i + N)
    {
        for (int j=0; j < N; j++){
            cout << fg[i+j].x << " ";
        }
        cout << endl;
    }
    cout << "----------------" << endl;
    for (int i = 0; i < N * N; i = i + N)
    {
        for (int j=0; j < N; j++){
            cout << fig[i+j].x << " ";
        }
        cout << endl;
    }
    cout << "----------------" << endl;

    int mem_size = sizeof(Complex)* SIZE;


    hipfftComplex *d_signal;
    checkCudaErrors(hipMalloc((void **) &d_signal, mem_size)); 
    checkCudaErrors(hipMemcpy(d_signal, fg, mem_size, hipMemcpyHostToDevice));

    hipfftComplex *d_filter_kernel;
    checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));
    checkCudaErrors(hipMemcpy(d_filter_kernel, fig, mem_size, hipMemcpyHostToDevice));

    // cout << d_signal[1].x << endl;
    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

    // Transform signal and filter
    printf("Transforming signal hipfftExecR2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel, (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD);

    printf("Launching Complex multiplication<<< >>>\n");
    ComplexMUL <<< N, N >> >(d_signal, d_filter_kernel);

    // Transform signal back
    printf("Transforming signal back hipfftExecC2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    Complex *result = new Complex[SIZE];
    hipMemcpy(result, d_signal, sizeof(Complex)*SIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i = i + N)
    {
        for (int j=0; j < N; j++){
            cout << result[i+j].x << " ";
        }
        cout << endl;
    }

    delete result, fg, fig;
    hipfftDestroy(plan);
    //hipfftDestroy(plan2);
    hipFree(d_signal);
    hipFree(d_filter_kernel);

}
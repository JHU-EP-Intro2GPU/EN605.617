
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#define HA 2
#define WA 9
#define WB 2
#define HB WA 
#define WC WB   
#define HC HA  
#define index(i,j,ld) (((j)*(ld))+(i))

void printMat(float*P,int uWP,int uHP){
  //printf("\n %f",P[1]);
  int i,j;
  for(i=0;i<uHP;i++){

      printf("\n");

      for(j=0;j<uWP;j++)
          printf("%f ",P[index(i,j,uHP)]);
  }
}




 int  main (int argc, char** argv) {
    hipblasStatus_t status;
    int i,j;
    cublasInit();

    float *A = (float*)malloc(HA*WA*sizeof(float));
    float *B = (float*)malloc(HB*WB*sizeof(float));
    float *C = (float*)malloc(HC*WC*sizeof(float));
    if (A == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
    }
    if (B == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
    }
    if (C == 0) {
      fprintf (stderr, "!!!! host memory allocation error (A)\n");
      return EXIT_FAILURE;
    }


    for (i=0;i<HA;i++)
      for (j=0;j<WA;j++)
        A[index(i,j,HA)] = (float) index(i,j,HA);   
    for (i=0;i<HB;i++)
      for (j=0;j<WB;j++)
        B[index(i,j,HB)] = (float) index(i,j,HB); 
    /*
    for (i=0;i<HA*WA;i++)
    A[i]=(float) i;
    for (i=0;i<HB*WB;i++)
    B[i]=(float) i;         */  


        float* AA; float* BB; float* CC;

    /*ALLOCATE ON THE DEVICE*/
    status=cublasAlloc(HA*WA,sizeof(float),(void**)&AA);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    status=cublasAlloc(HB*WB,sizeof(float),(void**)&BB);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    status=cublasAlloc(HC*WC,sizeof(float),(void**)&CC);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    /*SET MATRIX*/
    status=hipblasSetMatrix(HA,WA,sizeof(float),A,HA,AA,HA);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    status=hipblasSetMatrix(HB,WB,sizeof(float),B,HB,BB,HB);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device memory allocation error (A)\n");
      return EXIT_FAILURE;
    }

    /*KERNEL*/
    cublasSgemm('n','n',HA,WB,WA,1,AA,HA,BB,HB,0,CC,HC);

    status = cublasGetError();
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! kernel execution error.\n");
      return EXIT_FAILURE;
    }
    hipblasGetMatrix(HC,WC,sizeof(float),CC,HC,C,HC);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! device read error (A)\n");
      return EXIT_FAILURE;
    }


    /* PERFORMANCE OUTPUT*/

    printf("\nMatriz A:\n");
    printMat(A,WA,HA);
    printf("\nMatriz B:\n");
    printMat(B,WB,HB);
    printf("\nMatriz C:\n");
    printMat(C,WC,HC);

    free( A );  free( B );  free ( C );
    status = cublasFree(AA);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! memory free error (A)\n");
      return EXIT_FAILURE;
    }
    status = cublasFree(BB);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! memory free error (B)\n");
      return EXIT_FAILURE;
    }
    status = cublasFree(CC);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! memory free error (C)\n");
      return EXIT_FAILURE;
    }

    /* Shutdown */
    status = cublasShutdown();
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! shutdown error (A)\n");
      return EXIT_FAILURE;
    }

    if (argc > 1) {
      if (!strcmp(argv[1], "-noprompt") ||!strcmp(argv[1], "-qatest") ){
        return EXIT_SUCCESS;
      }
    } 
    else{
      printf("\nPress ENTER to exit...\n");
      getchar();
    }

		return EXIT_SUCCESS;


  }

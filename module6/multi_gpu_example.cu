#include "hip/hip_runtime.h"
#include "Utilities.cuh"
#include "InputOutput.cuh"

#define BLOCKSIZE 128

/*******************/
/* KERNEL FUNCTION */
/*******************/
template<class T>
__global__ void kernelFunction(T * __restrict__ d_data, const unsigned int NperGPU) {

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < NperGPU) for (int k = 0; k < 1000; k++) d_data[tid] = d_data[tid] * d_data[tid];

}

/******************/
/* PLAN STRUCTURE */
/******************/
template<class T>
struct plan {
    T *d_data;
};

/*********************/
/* SVD PLAN CREATION */
/*********************/
template<class T>
void createPlan(plan<T>& plan, unsigned int NperGPU, unsigned int gpuID) {

    // --- Device allocation
    gpuErrchk(hipSetDevice(gpuID));
    gpuErrchk(hipMalloc(&(plan.d_data), NperGPU * sizeof(T)));
}

/********/
/* MAIN */
/********/
int main() {

    const int numGPUs   = 4;
    const int NperGPU   = 500000;
    const int N         = NperGPU * numGPUs;

    plan<double> plan[numGPUs];
    for (int k = 0; k < numGPUs; k++) createPlan(plan[k], NperGPU, k);

    double *inputMatrices = (double *)malloc(N * sizeof(double));

    // --- "Breadth-first" approach - no async
    for (int k = 0; k < numGPUs; k++) {
        gpuErrchk(hipSetDevice(k));
        gpuErrchk(hipMemcpy(plan[k].d_data, inputMatrices + k * NperGPU, NperGPU * sizeof(double), hipMemcpyHostToDevice));
    }

    for (int k = 0; k < numGPUs; k++) {
        gpuErrchk(hipSetDevice(k));
        kernelFunction<<<iDivUp(NperGPU, BLOCKSIZE), BLOCKSIZE>>>(plan[k].d_data, NperGPU);
    }

    for (int k = 0; k < numGPUs; k++) {
        gpuErrchk(hipSetDevice(k));
        gpuErrchk(hipMemcpy(inputMatrices + k * NperGPU, plan[k].d_data, NperGPU * sizeof(double), hipMemcpyDeviceToHost));
    }

    gpuErrchk(hipDeviceReset());
}
/*https://cdac.in/index.aspx?id=ev_hpc_gpu-comp-nvidia-cuda-streams#hetr-cuda-prog-cuda-streams*/

#include <stdio.h> 
#include <time.h> 
#include <hip/hip_runtime.h> 

#define sizeOfArray 1024*1024
 
__global__ void arrayAddition(int *device_a, int *device_b, int *device_result)
{

	int threadId = threadIdx.x + blockIdx.x * blockDim.x ;

	if (threadId < sizeOfArray) 
        device_result[threadId]= device_a[threadId]+device_b[threadId]; 
} 


/* Check for safe return of all calls to the device */ 

int main ( int argc, char **argv ) 
{ 

  hipDeviceProp_t prop; 
  int *host_a, *host_b, *host_result; 
  int *device_a, *device_b, *device_result; 
  int whichDevice; 

  hipGetDeviceCount( &whichDevice); 
  hipGetDeviceProperties( &prop, whichDevice); 

  hipEvent_t start, stop; 
  float elapsedTime; 

  hipEventCreate( &start ); 
  hipEventCreate( &stop ); 

  hipStream_t stream; 
  hipStreamCreate(&stream); 

  hipMalloc( ( void**)& device_a, sizeOfArray * sizeof ( *device_a ) ); 
  hipMalloc( ( void**)& device_b,sizeOfArray * sizeof ( *device_b ) ); 
  hipMalloc( ( void**)& device_result, sizeOfArray * sizeof ( *device_result ) ); 

  hipHostAlloc((void **)&host_a, sizeOfArray*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&host_b, sizeOfArray*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&host_result, sizeOfArray*sizeof(int), hipHostMallocDefault);

  for(int index = 0; index < sizeOfArray; index++) 
  { 
   host_a[index] = rand()%10; 
   host_b[index] = rand()%10; 
  } 

  hipEventRecord(start);

  hipMemcpyAsync(device_a, host_a,sizeOfArray * sizeof ( int ), hipMemcpyHostToDevice, stream); 

  hipMemcpyAsync(device_b, host_b, sizeOfArray * sizeof ( int ), hipMemcpyHostToDevice, stream); 

  /*Kernel call*/ 

  arrayAddition <<<sizeOfArray, 1, 1, stream>>>(device_a, device_b, device_result);

  hipMemcpyAsync(host_result, device_result, sizeOfArray * sizeof ( int ), hipMemcpyDeviceToHost, stream);

  hipStreamSynchronize(stream);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  hipEventElapsedTime(&elapsedTime, start, stop); 

  printf("*********** CDAC - Tech Workshop : hyPACK-2013 \n"); 
  printf("\n Size of array : %d \n", sizeOfArray); 
  printf("\n Time taken: %3.1f ms \n", elapsedTime); 

  hipHostFree(host_a); 
  hipHostFree(host_b); 
  hipHostFree(host_result); 
  hipFree(device_a); 
  hipFree(device_b); 
  hipFree(device_result); 

  return 0; 
}

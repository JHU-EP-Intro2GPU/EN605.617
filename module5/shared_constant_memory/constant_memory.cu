#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define KERNEL_LOOP 65536

#define WORK_SIZE 256

typedef unsigned short int u16;
typedef unsigned int u32;

__constant__  static const unsigned int const_data_01 = 0x55555555;
__constant__  static const unsigned int const_data_02 = 0x77777777;
__constant__  static const unsigned int const_data_03 = 0x33333333;
__constant__  static const unsigned int const_data_04 = 0x11111111;

__global__ void const_test_gpu_literal(u32 * data,
		const u32 num_elements) {
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < num_elements) {
		u32 d = 0x55555555;

		for (int i = 0; i < KERNEL_LOOP; i++) {
			d ^= 0x55555555;
			d |= 0x77777777;
			d &= 0x33333333;
			d |= 0x11111111;
		}

		data[tid] = d;
	}
}

__global__ void const_test_gpu_const(unsigned int * const data, const unsigned int num_elements) {
	const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < num_elements) {
		unsigned int d = const_data_01;

		for (int i = 0; i < KERNEL_LOOP; i++) {
			d ^= const_data_01;
			d |= const_data_02;
			d &= const_data_03;
			d |= const_data_04;
		}

		data[tid] = d;
	}
}

__host__ void gpu_kernel(void) {
	const unsigned int num_elements = (128 * 1024);
	const unsigned int num_threads = 256;
	const unsigned int num_blocks = (num_elements + (num_threads - 1)) / num_threads;
	const unsigned int num_bytes = num_elements * sizeof(unsigned int);
	int max_device_num;
	const int max_runs = 6;

	hipGetDeviceCount(&max_device_num);

	for (int device_num = 0; device_num < max_device_num; device_num++) {
		hipSetDevice(device_num);

		for (int num_test = 0; num_test < max_runs; num_test++) {
			unsigned int * data_gpu;
			hipEvent_t kernel_start1, kernel_stop1;
			hipEvent_t kernel_start2, kernel_stop2;
			float delta_time1 = 0.0f, delta_time2 = 0.0F;
			struct hipDeviceProp_t device_prop;
			char device_prefix[261];

			hipMalloc(&data_gpu, num_bytes);
			hipEventCreate(&kernel_start1);
			hipEventCreate(&kernel_start2);
			
					hipEventCreateWithFlags(&kernel_stop1,
							hipEventBlockingSync);
			
					hipEventCreateWithFlags(&kernel_stop2,
							hipEventBlockingSync);

			hipGetDeviceProperties(&device_prop, device_num);
			sprintf(device_prefix, "ID: %d %s:", device_num, device_prop.name);

			const_test_gpu_literal<<<num_blocks, num_threads>>>(data_gpu,
					num_elements);

//			cuda_error_check("Error ",
//					" returned from literal startup  kernel!");

			hipEventRecord(kernel_start1, 0);
			const_test_gpu_literal<<<num_blocks, num_threads>>>(data_gpu,
					num_elements);

//			cuda_error_check("Error ",
//					" returned from literal runtime  kernel!");

			hipEventRecord(kernel_stop1, 0);
			hipEventSynchronize(kernel_stop1);
			
					hipEventElapsedTime(&delta_time1, kernel_start1,
							kernel_stop1);

			const_test_gpu_const<<<num_blocks, num_threads>>>(data_gpu,
					num_elements);

//			cuda_error_check("Error ",
//					" returned from literal startup  kernel!");

			hipEventRecord(kernel_stop2, 0);
			hipEventSynchronize(kernel_stop2);
			
					hipEventElapsedTime(&delta_time2, kernel_start2,
							kernel_stop2);

			if (delta_time1 > delta_time2) {
				printf(
						"\n%sConstant version is faster by: %.2fms (Const=%.2fms vs. Literal=%.2fms)",
						device_prefix, delta_time1 - delta_time2, delta_time1,
						delta_time2);
			} else {
				printf(
						"\n%sLiteral version is faster by: %.2fms (Const=%.2fms vs. Literal=%.2fms)",
						device_prefix, delta_time2 - delta_time1, delta_time1,
						delta_time2);
			}

			hipEventDestroy(kernel_start1);
			hipEventDestroy(kernel_start2);
			hipEventDestroy(kernel_stop1);
			hipEventDestroy(kernel_stop2);
			hipFree(data_gpu);
		}

		hipDeviceReset();
		printf("\n");
	}
//	wait_exit();
}

__device__  static unsigned int data_01 = 0x55555555;
__device__  static unsigned int data_02 = 0x77777777;
__device__  static unsigned int data_03 = 0x33333333;
__device__  static unsigned int data_04 = 0x11111111;

__global__ void const_test_gpu_gmem(unsigned int * const data, const unsigned int num_elements) {
	const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (tid < num_elements) {
		unsigned int d = data_01;

		for (int i = 0; i < KERNEL_LOOP; i++) {
			d ^= data_01;
			d |= data_02;
			d &= data_03;
			d |= data_04;
		}

		data[tid] = d;
	}
}

void execute_host_functions()
{

}

void execute_gpu_functions()
{
	u32 *data = NULL;
	const u32 num_threads = 256;
	const u32 num_blocks = WORK_SIZE/num_threads;

	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];
	int i;
	for (i = 0; i < WORK_SIZE; i++){
		idata[i] = (unsigned int) i;
	}

	hipMalloc((void** ) &data, sizeof(int) * WORK_SIZE);
	
	hipMemcpy(data, idata, sizeof(unsigned int) * WORK_SIZE, hipMemcpyHostToDevice);

	const_test_gpu_literal<<<num_blocks,num_threads>>>(data, WORK_SIZE);
	hipDeviceSynchronize();	// Wait for the GPU launched work to complete
	hipGetLastError();
	
	hipMemcpy(odata, data, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost);

	for (i = 0; i < WORK_SIZE; i++) {
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);
	}
	
	hipFree((void* ) data);
	hipDeviceReset();
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	execute_host_functions();
	execute_gpu_functions();

	return 0;
}

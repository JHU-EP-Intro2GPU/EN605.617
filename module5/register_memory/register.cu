#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define KERNEL_LOOP 2048
#define KERNEL_SIZE 128

__host__ void wait_exit(void)
{
        char ch;

        printf("\nPress any key to exit");
        ch = getchar();
}

__host__ void generate_rand_data(unsigned int * host_data_ptr)
{
        for(unsigned int i=0; i < KERNEL_LOOP; i++)
        {
                host_data_ptr[i] = (unsigned int) rand();
        }
}

__global__ void test_gpu_register(unsigned int * const data, const unsigned int num_elements)
{
        const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if(tid < num_elements)
        {
                unsigned int d_tmp = data[tid];
                d_tmp = d_tmp * 2;
                data[tid] = d_tmp;
        }
}

__host__ void gpu_kernel(void)
{
        const unsigned int num_elements = KERNEL_LOOP;
        const unsigned int num_threads = KERNEL_SIZE;
        const unsigned int num_blocks = (num_elements + num_threads - 1)/num_threads;
        const unsigned int num_bytes = num_elements * sizeof(unsigned int);

        unsigned int * data_gpu;

        unsigned int host_packed_array[num_elements];
        unsigned int host_packed_array_output[num_elements];

        hipMalloc(&data_gpu, num_bytes);

        generate_rand_data(host_packed_array);

        hipMemcpy(data_gpu, host_packed_array, num_bytes,hipMemcpyHostToDevice);

        test_gpu_register <<<num_blocks, num_threads>>>(data_gpu, num_elements);

        hipDeviceSynchronize();        // Wait for the GPU launched work to complete
        hipGetLastError();

        hipMemcpy(host_packed_array_output, data_gpu, num_bytes,hipMemcpyDeviceToHost);

        for (int i = 0; i < num_elements; i++){
                printf("Input value: %x, device output: %x\n",host_packed_array[i], host_packed_array_output[i]);
        }

        hipFree((void* ) data_gpu);
        hipDeviceReset();
//        wait_exit();
}

void execute_host_functions()
{

}

void execute_gpu_functions()
{
	gpu_kernel();
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	execute_host_functions();
	execute_gpu_functions();

	return EXIT_SUCCESS;
}
